#include "hip/hip_runtime.h"
//Names: Ruben and Abdul
//CPU: AMD Ryzen 7 7735HS
// g++ main_threading.cpp -o ./a.out -O3 -lwebp -lpng
//(O3 Optimization enabled)
/*
mandelbrot duration: 17.9092 seconds 
*/


//GPU: NVIDIA RTX 4060 LAPTOP GPU
// nvcc main_cuda.cu -o ./a.out -O3 -lwebp -lpng
/*
mandelbrot duration: 1.61442 seconds
*/


#include <cstdint>
#include <cmath>
#include <complex>
#include <iostream>
#include <fstream>
#include <webp/encode.h>
#include <png.h>
#include <chrono>
#include <thread>
#include <vector> 
#include <hip/hip_runtime.h>



/*
    C = (0,0)
    f(z) = z^2 + C 
            (0,0) + (0,0)

    C = (2,0)
    f(z) = z^2 + C
            (4,0) + (2,0) = (6,0)
            (36,0) + (6,0)...


            with vecorized coding, if statements are a problem

            because all numbers in the vector are processed the same
*/
using namespace std;



//Use the __managed__ keyword to declare this array as being shared between host and device
__managed__ uint32_t* count_arr;


//Cuda does not support complex number handling
__device__ float complex_abs(float x, float y) {
    return sqrtf(x * x + y * y);
}

__device__ float mag_sqr(float x, float y) {
    return x * x + y * y;
}


__global__ void mandelbrot_worker(uint32_t* count_arr, uint32_t w, uint32_t h,
                                  const uint32_t max_count, const float xmin,
                                  const float xmax, const float ymin, const float ymax) {
    // Calculate global thread coordinates in the 2D Mandelbrot set grid
    int j = blockIdx.x * blockDim.x + threadIdx.x;  // x-coordinate
    int i = blockIdx.y * blockDim.y + threadIdx.y;  // y-coordinate

    // if (i >= h || j >= w) {
    //     printf("ERROR: OUT OF BOUNDS");
    //     return;
    // } // Bounds check

    // Map pixel coordinate (i, j) to complex plane coordinates (x, y)
    float x0 = xmin + (xmax - xmin) * j / w;
    float y0 = ymin + (ymax - ymin) * i / h;

    // Compute Mandelbrot iteration count for point (x0, y0)
    float x = 0.0f, y = 0.0f;
    uint32_t count = 0;

    while (count < max_count && complex_abs(x, y) <= 2.0f) {
        float x_temp = x * x - y * y + x0;
        y = 2.0f * x * y + y0;
        x = x_temp;
        count++;
    }

    // Write result to the output array
    count_arr[i * w + j] = count;
}

__global__ void mandelbrot_worker2(uint32_t* count_arr, uint32_t w, uint32_t h,
                                  const uint32_t max_count, const float xmin,
                                  const float dx_per_pixel, const float ymin, const float dy_per_pixel) {
    // Calculate global thread coordinates in the 2D Mandelbrot set grid
    int j = blockIdx.x * blockDim.x + threadIdx.x;  // x-coordinate
    int i = blockIdx.y * blockDim.y + threadIdx.y;  // y-coordinate

    // if (i >= h || j >= w) {
    //     printf("ERROR: OUT OF BOUNDS");
    //     return;
    // } // Bounds check

    // Map pixel coordinate (i, j) to complex plane coordinates (x, y)
    float x0 = xmin + dx_per_pixel * j ;
    float y0 = ymin + dy_per_pixel * i ;

    // Compute Mandelbrot iteration count for point (x0, y0)
    float x = 0.0f, y = 0.0f;
    uint32_t count = 0;

    while (count < max_count && mag_sqr(x, y) <= 4.0f) {
        float x_temp = x * x - y * y + x0;
        y = 2.0f * x * y + y0;
        x = x_temp;
        count++;
    }

    // Write result to the output array
    count_arr[i * w + j] = count;
}



void mandelbrot(uint32_t count_arr[], uint32_t w, uint32_t h,const uint32_t max_count, const float xmin, const float xmax, const float ymin, const float ymax){
    // sequentially write each count to array
    uint32_t num_threads = 16;
    if(num_threads%4!=0 && num_threads!=1){
        return;
    }
    // Launch kernel
    dim3 threadsPerBlock(num_threads, num_threads);  // 16x16 block of threads
    dim3 numBlocks((w + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (h + threadsPerBlock.y - 1) / threadsPerBlock.y);


    const float dx_per_pixel = ((xmax - xmin)/ w)  ;
    const float dy_per_pixel = ((ymax - ymin)/ h)  ;
    // mandelbrot_worker<<<numBlocks, threadsPerBlock>>>(count_arr, w, h, max_count, xmin, xmax, ymin, ymax);
    mandelbrot_worker2<<<numBlocks, threadsPerBlock>>>(count_arr, w, h, max_count, xmin, dx_per_pixel, ymin, dy_per_pixel);

    hipDeviceSynchronize();//Wait for GPU code
}

void convert_mandelbrot_count_to_rgb(uint32_t pixels[], uint32_t mandelbrot_count[], uint32_t w, uint32_t h, const uint32_t colors[], uint32_t color_count) {
    for (uint32_t y = 0; y < h; y++) {
        for (uint32_t x = 0; x < w; x++) {
            uint32_t index = y * w + x;
            uint32_t count_value = mandelbrot_count[index];

            // Normalize the Mandelbrot iteration count and map it to a color
            uint32_t color_index = count_value % color_count;  // Cyclic mapping if count > color_count
            pixels[index] = colors[color_index];
        }
    }
}

void build_color_table(uint32_t colors[], uint32_t count) {
    for (uint32_t i = 0; i < count; i++) {
        // Generate a color based on the position in the palette
        uint8_t r = (i * 5) % 255;  // Adjust values to create a gradient
        uint8_t g = (i * 7) % 255;  // Feel free to tweak the multipliers
        uint8_t b = (i * 11) % 255; // to achieve different patterns
        uint8_t a = 0xFF;           // Set transparency to opaque

        // Combine color components into a single 32-bit value
        colors[i] = (a << 24) | (r << 16) | (g << 8) | b;
    }
}

bool save_webp(const char* filename, uint32_t* pixels, uint32_t w, uint32_t h, int quality) {
    // Convert the array of pixels (in RGBA format) to a WebP-encoded buffer
    uint8_t* webp_data;
    size_t webp_size = WebPEncodeRGBA((uint8_t*)pixels, w, h, w * 4, quality, &webp_data);
    
    if (webp_size == 0) {
        std::cerr << "Error encoding WebP image!" << std::endl;
        return false; // Encoding failed
    }

    // Save the WebP-encoded buffer to a file
    std::ofstream file(filename, std::ios::binary);
    if (!file) {
        std::cerr << "Error opening file for writing!" << std::endl;
        WebPFree(webp_data); // Free the WebP data in case of error
        return false;
    }

    file.write(reinterpret_cast<const char*>(webp_data), webp_size);
    file.close();
    
    // Free the WebP buffer allocated by WebPEncodeRGBA
    WebPFree(webp_data);

    return true;
}
void write_png(const char* filename, uint32_t* pixels, int w, int h) {
    FILE *fp = fopen(filename, "wb");
    if(!fp) abort();

    png_structp png = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png) abort();

    png_infop info = png_create_info_struct(png);
    if (!info) abort();

    if (setjmp(png_jmpbuf(png))) abort();

    png_init_io(png, fp);

    // Output is 8bit depth, RGBA format.
    png_set_IHDR(
      png,
      info,
      w, h,
      8,
      PNG_COLOR_TYPE_RGBA,
      PNG_INTERLACE_NONE,
      PNG_COMPRESSION_TYPE_DEFAULT,
      PNG_FILTER_TYPE_DEFAULT
    );
    png_write_info(png, info);

    // To write image data
    png_bytep row = (png_bytep) malloc(4 * w * sizeof(png_byte));
    for(int y = 0; y < h; y++) {
        for(int x = 0; x < w; x++) {
            uint32_t pixel = pixels[y * w + x];
            png_bytep color = &(row[x * 4]);
            color[0] = (pixel >> 16) & 0xFF; // Red
            color[1] = (pixel >> 8) & 0xFF;  // Green
            color[2] = pixel & 0xFF;         // Blue
            color[3] = (pixel >> 24) & 0xFF; // Alpha
        }
        png_write_row(png, row);
    }
    png_write_end(png, NULL);

    fclose(fp);
    png_free_data(png, info, PNG_FREE_ALL, -1);
    png_destroy_write_struct(&png, &info);
    free(row);
}


int main() {
    const long w = 3840*8;  // Width for 4K * 4
    const long h = 2160*8;  // Height for 4K * 4
    uint32_t colors[64];
    uint32_t* pixels = new uint32_t[w * h];

    // Allocate managed memory
    hipMallocManaged(&count_arr, w * h * sizeof(uint32_t));

    // Build color table
    auto start = std::chrono::high_resolution_clock::now();
    build_color_table(colors, 64);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> build_color_table_duration = end - start;
    std::cout << "build_color_table duration: " << build_color_table_duration.count() << " seconds\n";

    // Generate Mandelbrot set counts
    start = std::chrono::high_resolution_clock::now();
    mandelbrot(count_arr, w, h, 64, -2, 0.47, -1.12, 1.12);
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> mandelbrot_duration = end - start;
    std::cout << "mandelbrot duration: " << mandelbrot_duration.count() << " seconds\n";
    
    // Convert counts to RGB values
    start = std::chrono::high_resolution_clock::now();
    convert_mandelbrot_count_to_rgb(pixels, count_arr, w, h, colors, 64);
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> convert_to_rgb_duration = end - start;
    std::cout << "convert_mandelbrot_count_to_rgb duration: " << convert_to_rgb_duration.count() << " seconds\n";

    // Save as WebP
    start = std::chrono::high_resolution_clock::now();
    write_png("out.png", pixels, w, h);
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> write_png_duration = end - start;
    std::cout << "write_png duration: " << write_png_duration.count() << " seconds\n";
    save_webp("out.webp", pixels, w, h,100);

    hipFree(count_arr);
    delete[] pixels;
}